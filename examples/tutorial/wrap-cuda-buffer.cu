#include "hip/hip_runtime.h"
/* =========================================================================
   Copyright (c) 2010-2013, Institute for Microelectronics,
                            Institute for Analysis and Scientific Computing,
                            TU Wien.
   Portions of this software are copyright by UChicago Argonne, LLC.

                            -----------------
                  ViennaCL - The Vienna Computing Library
                            -----------------

   Project Head:    Karl Rupp                   rupp@iue.tuwien.ac.at

   (A list of authors and contributors can be found in the PDF manual)

   License:         MIT (X11), see file LICENSE in the base directory
============================================================================= */

/*
*
*   Tutorial:  Use ViennaCL within user-defined (i.e. your own) OpenCL contexts
*
*/


//
// include necessary system headers
//
#include <iostream>
#include <string>

#include <hip/hip_runtime.h>

//
// ViennaCL includes
//
#include "viennacl/vector.hpp"
#include "viennacl/matrix.hpp"
#include "viennacl/linalg/matrix_operations.hpp"
#include "viennacl/linalg/norm_2.hpp"
#include "viennacl/linalg/prod.hpp"


// Some helper functions for this tutorial:
#include "Random.hpp"

template <typename T>
__global__ void my_inplace_add_kernel(T * vec1, T * vec2, unsigned int size)
{
    for (unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
                      i < size;
                      i += gridDim.x * blockDim.x)
      vec1[i] += vec2[i];
}



int main()
{
  typedef float       ScalarType;

  //
  // Part 1: Allocate some CUDA memory
  //
  std::size_t size = 10;
  ScalarType *cuda_x;
  ScalarType *cuda_y;

  hipMalloc(&cuda_x, size * sizeof(ScalarType));
  hipMalloc(&cuda_y, size * sizeof(ScalarType));

  std::vector<ScalarType> host_x(size, 1.0);
  std::vector<ScalarType> host_y(size, 2.0);

  hipMemcpy(cuda_x, &(host_x[0]), size * sizeof(ScalarType), hipMemcpyHostToDevice);
  hipMemcpy(cuda_y, &(host_y[0]), size * sizeof(ScalarType), hipMemcpyHostToDevice);

  my_inplace_add_kernel<<<128, 128>>>(cuda_x, cuda_y, static_cast<unsigned int>(1000));

  // check result
  std::vector<ScalarType> result_cuda(size);
  hipMemcpy(&(result_cuda[0]), cuda_x, size * sizeof(ScalarType), hipMemcpyDeviceToHost);

  std::cout << "result_cuda: ";
  for (std::size_t i=0; i<size; ++i)
    std::cout << result_cuda[i] << " ";
  std::cout << std::endl;

  //
  // Part 2: Now do the same within ViennaCL
  //

  viennacl::vector<ScalarType> vcl_vec1(cuda_x, size, viennacl::CUDA_MEMORY); // Third parameter specifies that this is CUDA memory rather than host memory
  viennacl::vector<ScalarType> vcl_vec2(cuda_y, size, viennacl::CUDA_MEMORY); // Third parameter specifies that this is CUDA memory rather than host memory

  vcl_vec1 = viennacl::scalar_vector<ScalarType>(size, ScalarType(1.0));
  vcl_vec2 = viennacl::scalar_vector<ScalarType>(size, ScalarType(2.0));

  vcl_vec1 += vcl_vec2;

  std::cout << "Result with ViennaCL: ";
  std::cout << vcl_vec1 << std::endl;

  //
  //  That's it.
  //
  std::cout << "!!!! TUTORIAL COMPLETED SUCCESSFULLY !!!!" << std::endl;

  return 0;
}


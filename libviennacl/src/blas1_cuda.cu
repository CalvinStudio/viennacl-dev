#include "hip/hip_runtime.h"
/* =========================================================================
   Copyright (c) 2010-2013, Institute for Microelectronics,
                            Institute for Analysis and Scientific Computing,
                            TU Wien.
   Portions of this software are copyright by UChicago Argonne, LLC.

                            -----------------
                  ViennaCL - The Vienna Computing Library
                            -----------------

   Project Head:    Karl Rupp                   rupp@iue.tuwien.ac.at

   (A list of authors and contributors can be found in the PDF manual)

   License:         MIT (X11), see file LICENSE in the base directory
============================================================================= */

// include necessary system headers
#include <iostream>

#include "viennacl.hpp"
#include "viennacl_private.hpp"

//include basic scalar and vector types of ViennaCL
#include "viennacl/scalar.hpp"
#include "viennacl/vector.hpp"

//include the generic inner product functions of ViennaCL
#include "viennacl/linalg/inner_prod.hpp"

//include the generic norm functions of ViennaCL
#include "viennacl/linalg/norm_1.hpp"
#include "viennacl/linalg/norm_2.hpp"
#include "viennacl/linalg/norm_inf.hpp"


#ifdef VIENNACL_WITH_CUDA


// IxAMAX

ViennaCLStatus ViennaCLCUDAiSamax(ViennaCLBackend /*backend*/, ViennaCLInt n,
                                  ViennaCLInt *index,
                                  float *x, ViennaCLInt offx, ViennaCLInt incx)
{
  viennacl::vector_base<float> v1(x, viennacl::CUDA_MEMORY, n, offx, incx);

  *index = viennacl::linalg::index_norm_inf(v1);
  return ViennaCLSuccess;
}

ViennaCLStatus ViennaCLCUDAiDamax(ViennaCLBackend /*backend*/, ViennaCLInt n,
                                  ViennaCLInt *index,
                                  double *x, ViennaCLInt offx, ViennaCLInt incx)
{
  viennacl::vector_base<double> v1(x, viennacl::CUDA_MEMORY, n, offx, incx);

  *index = viennacl::linalg::index_norm_inf(v1);
  return ViennaCLSuccess;
}



// xASUM

ViennaCLStatus ViennaCLCUDASasum(ViennaCLBackend /*backend*/, ViennaCLInt n,
                                 float *alpha,
                                 float *x, ViennaCLInt offx, ViennaCLInt incx)
{
  viennacl::vector_base<float> v1(x, viennacl::CUDA_MEMORY, n, offx, incx);

  *alpha = viennacl::linalg::norm_1(v1);
  return ViennaCLSuccess;
}

ViennaCLStatus ViennaCLCUDADasum(ViennaCLBackend /*backend*/, ViennaCLInt n,
                                 double *alpha,
                                 double *x, ViennaCLInt offx, ViennaCLInt incx)
{
  viennacl::vector_base<double> v1(x, viennacl::CUDA_MEMORY, n, offx, incx);

  *alpha = viennacl::linalg::norm_1(v1);
  return ViennaCLSuccess;
}


// xAXPY

ViennaCLStatus ViennaCLCUDASaxpy(ViennaCLBackend /*backend*/, ViennaCLInt n,
                                 float alpha,
                                 float *x, ViennaCLInt offx, ViennaCLInt incx,
                                 float *y, ViennaCLInt offy, ViennaCLInt incy)
{
  viennacl::vector_base<float> v1(x, viennacl::CUDA_MEMORY, n, offx, incx);
  viennacl::vector_base<float> v2(y, viennacl::CUDA_MEMORY, n, offy, incy);

  v2 += alpha * v1;
  return ViennaCLSuccess;
}

ViennaCLStatus ViennaCLCUDADaxpy(ViennaCLBackend /*backend*/, ViennaCLInt n,
                                 double alpha,
                                 double *x, ViennaCLInt offx, ViennaCLInt incx,
                                 double *y, ViennaCLInt offy, ViennaCLInt incy)
{
  viennacl::vector_base<double> v1(x, viennacl::CUDA_MEMORY, n, offx, incx);
  viennacl::vector_base<double> v2(y, viennacl::CUDA_MEMORY, n, offy, incy);

  v2 += alpha * v1;
  return ViennaCLSuccess;
}


// xCOPY

ViennaCLStatus ViennaCLCUDAScopy(ViennaCLBackend /*backend*/, ViennaCLInt n,
                                 float *x, ViennaCLInt offx, ViennaCLInt incx,
                                 float *y, ViennaCLInt offy, ViennaCLInt incy)
{
  viennacl::vector_base<float> v1(x, viennacl::CUDA_MEMORY, n, offx, incx);
  viennacl::vector_base<float> v2(y, viennacl::CUDA_MEMORY, n, offy, incy);

  v2 = v1;
  return ViennaCLSuccess;
}

ViennaCLStatus ViennaCLCUDADcopy(ViennaCLBackend /*backend*/, ViennaCLInt n,
                                 double *x, ViennaCLInt offx, ViennaCLInt incx,
                                 double *y, ViennaCLInt offy, ViennaCLInt incy)
{
  viennacl::vector_base<double> v1(x, viennacl::CUDA_MEMORY, n, offx, incx);
  viennacl::vector_base<double> v2(y, viennacl::CUDA_MEMORY, n, offy, incy);

  v2 = v1;
  return ViennaCLSuccess;
}

// xDOT

ViennaCLStatus ViennaCLCUDASdot(ViennaCLBackend /*backend*/, ViennaCLInt n,
                                float *alpha,
                                float *x, ViennaCLInt offx, ViennaCLInt incx,
                                float *y, ViennaCLInt offy, ViennaCLInt incy)
{
  viennacl::vector_base<float> v1(x, viennacl::CUDA_MEMORY, n, offx, incx);
  viennacl::vector_base<float> v2(y, viennacl::CUDA_MEMORY, n, offy, incy);

  *alpha = viennacl::linalg::inner_prod(v1, v2);
  return ViennaCLSuccess;
}

ViennaCLStatus ViennaCLCUDADdot(ViennaCLBackend /*backend*/, ViennaCLInt n,
                                double *alpha,
                                double *x, ViennaCLInt offx, ViennaCLInt incx,
                                double *y, ViennaCLInt offy, ViennaCLInt incy)
{
  viennacl::vector_base<double> v1(x, viennacl::CUDA_MEMORY, n, offx, incx);
  viennacl::vector_base<double> v2(y, viennacl::CUDA_MEMORY, n, offy, incy);

  *alpha = viennacl::linalg::inner_prod(v1, v2);
  return ViennaCLSuccess;
}

// xNRM2

ViennaCLStatus ViennaCLCUDASnrm2(ViennaCLBackend /*backend*/, ViennaCLInt n,
                                 float *alpha,
                                 float *x, ViennaCLInt offx, ViennaCLInt incx)
{
  viennacl::vector_base<float> v1(x, viennacl::CUDA_MEMORY, n, offx, incx);

  *alpha = viennacl::linalg::norm_2(v1);
  return ViennaCLSuccess;
}

ViennaCLStatus ViennaCLCUDADnrm2(ViennaCLBackend /*backend*/, ViennaCLInt n,
                                 double *alpha,
                                 double *x, ViennaCLInt offx, ViennaCLInt incx)
{
  viennacl::vector_base<double> v1(x, viennacl::CUDA_MEMORY, n, offx, incx);

  *alpha = viennacl::linalg::norm_2(v1);
  return ViennaCLSuccess;
}



// xROT

ViennaCLStatus ViennaCLCUDASrot(ViennaCLBackend /*backend*/, ViennaCLInt n,
                                float *x, ViennaCLInt offx, ViennaCLInt incx,
                                float *y, ViennaCLInt offy, ViennaCLInt incy,
                                float c, float s)
{
  viennacl::vector_base<float> v1(x, viennacl::CUDA_MEMORY, n, offx, incx);
  viennacl::vector_base<float> v2(y, viennacl::CUDA_MEMORY, n, offy, incy);

  viennacl::linalg::plane_rotation(v1, v2, c, s);
  return ViennaCLSuccess;
}

ViennaCLStatus ViennaCLCUDADrot(ViennaCLBackend /*backend*/, ViennaCLInt n,
                                double *x, ViennaCLInt offx, ViennaCLInt incx,
                                double *y, ViennaCLInt offy, ViennaCLInt incy,
                                double c, double s)
{
  viennacl::vector_base<double> v1(x, viennacl::CUDA_MEMORY, n, offx, incx);
  viennacl::vector_base<double> v2(y, viennacl::CUDA_MEMORY, n, offy, incy);

  viennacl::linalg::plane_rotation(v1, v2, c, s);
  return ViennaCLSuccess;
}



// xSCAL

ViennaCLStatus ViennaCLCUDASscal(ViennaCLBackend /*backend*/, ViennaCLInt n,
                                 float alpha,
                                 float *x, ViennaCLInt offx, ViennaCLInt incx)
{
  viennacl::vector_base<float> v1(x, viennacl::CUDA_MEMORY, n, offx, incx);

  v1 *= alpha;
  return ViennaCLSuccess;
}

ViennaCLStatus ViennaCLCUDADscal(ViennaCLBackend /*backend*/, ViennaCLInt n,
                                 double alpha,
                                 double *x, ViennaCLInt offx, ViennaCLInt incx)
{
  viennacl::vector_base<double> v1(x, viennacl::CUDA_MEMORY, n, offx, incx);

  v1 *= alpha;
  return ViennaCLSuccess;
}


// xSWAP

ViennaCLStatus ViennaCLCUDASswap(ViennaCLBackend /*backend*/, ViennaCLInt n,
                                 float *x, ViennaCLInt offx, ViennaCLInt incx,
                                 float *y, ViennaCLInt offy, ViennaCLInt incy)
{
  viennacl::vector_base<float> v1(x, viennacl::CUDA_MEMORY, n, offx, incx);
  viennacl::vector_base<float> v2(y, viennacl::CUDA_MEMORY, n, offy, incy);

  viennacl::swap(v1, v2);
  return ViennaCLSuccess;
}

ViennaCLStatus ViennaCLCUDADswap(ViennaCLBackend /*backend*/, ViennaCLInt n,
                                 double *x, ViennaCLInt offx, ViennaCLInt incx,
                                 double *y, ViennaCLInt offy, ViennaCLInt incy)
{
  viennacl::vector_base<double> v1(x, viennacl::CUDA_MEMORY, n, offx, incx);
  viennacl::vector_base<double> v2(y, viennacl::CUDA_MEMORY, n, offy, incy);

  viennacl::swap(v1, v2);
  return ViennaCLSuccess;
}
#endif


